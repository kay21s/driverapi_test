
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void matSum(int *a, int i)
{
	int tid = blockIdx.x;
	if (threadIdx.x == 0) printf("my block id is %d, a is %d\n", tid, *a);

	clock_t start = clock();
	clock_t now;

		printf("i is %d\n", i);
		return;

	for (;;) {
		now = clock();
		clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
		if (cycles >= 100000) {
			printf("A is %d\n", *a);
			start = clock();
			//break;
		}
	}
}
